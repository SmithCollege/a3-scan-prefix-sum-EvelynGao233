#include <stdio.h>
#include <hip/hip_runtime.h>

int sizes[] = {100, 1000, 10000, 100000, 1000000, 10000000, 100000000};

// naive
__global__ void naivePrefixScan(int* input, int* output, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index >= size) return;

    int value = 0;

    for (int j = 0; j <= index; j++) {
        value += input[j];
    }

    output[index] = value;
}

// recursive doubling
__global__ void recursivePrefixScan(int* input, int* output, int size) {
    extern __shared__ int temp[];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index >= size) return;

    temp[threadIdx.x] = input[index];
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int tempVal = 0;
        if (threadIdx.x >= stride) {
            tempVal = temp[threadIdx.x - stride];
        }
        __syncthreads();
        temp[threadIdx.x] += tempVal;
        __syncthreads();
    }

    output[index] = temp[threadIdx.x];
}

int main() {
    for (int s = 0; s < 7; s++) {
        int SIZE = sizes[s];

        int* input;
        int* output;

        hipMallocManaged(&input, SIZE * sizeof(int));
        hipMallocManaged(&output, SIZE * sizeof(int));

        for (int i = 0; i < SIZE; i++) {
            input[i] = 1;
        }

        int threadsPerBlock = (SIZE < 256) ? SIZE : 256;
        int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;

        // learned from cuda post: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // naivePrefixScan<<<blocksPerGrid, threadsPerBlock>>>(input, output, SIZE);
        recursivePrefixScan<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(input, output, SIZE);

        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Print time taken for each size
        printf("Size: %d, Time taken (GPU): %f ms\n", SIZE, milliseconds);

        // Print first 10 results for the largest size only
        if (s == 6) {
            for (int i = 0; i < 10; i++) {
                printf("%d ", output[i]);
            }
            printf("\n");
        }

        // Free memory
        hipFree(input);
        hipFree(output);
    }

    return 0;
}
